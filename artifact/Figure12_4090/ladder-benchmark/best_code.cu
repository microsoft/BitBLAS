
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(128) Fused(float* __restrict__ A, int8_t* __restrict__ B, uint8_t* __restrict__ Scales, float* __restrict__ C) {
  
  float in_thread_C_local[1];
  float A_local[4];
  signed char B_local[4];
  float B_decode_local[4];
  __shared__ float red_buf0[128];
  in_thread_C_local[0] = 0.000000e+00f;
  for (int k_0 = 0; k_0 < 56; ++k_0) {
    *(float4*)(A_local + 0) = *(float4*)(A + ((k_0 * 512) + (((int)threadIdx.x) * 4)));
    *(int*)(B_local + 0) = *(int*)(B + (((((int)blockIdx.x) * 28672) + (k_0 * 512)) + (((int)threadIdx.x) * 4)));
    for (int ax1 = 0; ax1 < 4; ++ax1) {
        uint __1 = (((max((((((((uint)B_local[ax1]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) + ((uint)Scales[(((k_0 * 131072) + ((((int)threadIdx.x) >> 3) * 8192)) + ((int)blockIdx.x))])), (uint)63) | ((((((uint)B_local[ax1]) >> (uint)0) & (uint)255) >> (uint)7) << (uint)8)) << (uint)2) | (((((((uint)B_local[ax1]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) & (uint)2)) << (uint)25;
      B_decode_local[ax1] = (*(float *)(&(__1)));
    }
    for (int k_2_0 = 0; k_2_0 < 2; ++k_2_0) {
      for (int k_2_1 = 0; k_2_1 < 2; ++k_2_1) {
        in_thread_C_local[0] = (in_thread_C_local[0] + (A_local[((k_2_0 * 2) + k_2_1)] * B_decode_local[((k_2_0 * 2) + k_2_1)]));
      }
    }
  }
  __syncthreads();
  ((volatile float*)red_buf0)[((int)threadIdx.x)] = in_thread_C_local[0];
  __syncthreads();
  if (((int)threadIdx.x) < 64) {
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = (((volatile float*)red_buf0)[((int)threadIdx.x)] + ((volatile float*)red_buf0)[(((int)threadIdx.x) + 64)]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = (((volatile float*)red_buf0)[((int)threadIdx.x)] + ((volatile float*)red_buf0)[(((int)threadIdx.x) + 32)]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    float w_16_0 = (((volatile float*)red_buf0)[((int)threadIdx.x)] + ((volatile float*)red_buf0)[(((int)threadIdx.x) + 16)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_16_0;
    float w_8_0 = (((volatile float*)red_buf0)[((int)threadIdx.x)] + ((volatile float*)red_buf0)[(((int)threadIdx.x) + 8)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_8_0;
    float w_4_0 = (((volatile float*)red_buf0)[((int)threadIdx.x)] + ((volatile float*)red_buf0)[(((int)threadIdx.x) + 4)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_4_0;
    float w_2_0 = (((volatile float*)red_buf0)[((int)threadIdx.x)] + ((volatile float*)red_buf0)[(((int)threadIdx.x) + 2)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_2_0;
    float w_1_0 = (((volatile float*)red_buf0)[((int)threadIdx.x)] + ((volatile float*)red_buf0)[(((int)threadIdx.x) + 1)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_1_0;
  }
  __syncthreads();
  C[((int)blockIdx.x)] = ((volatile float*)red_buf0)[0];
}

